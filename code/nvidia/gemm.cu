#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <memory>
#include <stdexcept>
#include <vector>
#include <tuple>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>


template<typename T>
class Matrix {
public:
    Matrix(int row, int col):n_row(row), n_col(col), size(row*col) {
        T *tmp;
        if (hipMalloc(&tmp, size * sizeof(T)) != hipSuccess) {
            printf("row:%d, col:%d, size:%d\n", row, col, size);
            throw std::bad_alloc();
        }
        data_ptr.reset(tmp, [](T *p){ hipFree(p); });
    }

    void fillRand(hiprandGenerator_t curand_gen) {
        hiprandGenerateUniform(curand_gen, data(), size);    
    }

    void fill(T val) {
        thrust::fill(thrust::device_ptr<T>(data()),
                     thrust::device_ptr<T>(data() + size),
                     val);
    }

    int row() { return n_row; }
    int col() { return n_col; }
    T* data() { return data_ptr.get(); }

private:
    int n_row, n_col, size;
    std::shared_ptr<T> data_ptr;
};





double gemm(Matrix<float> A, Matrix<float> B, Matrix<float> C, bool at, bool bt, hipblasHandle_t cublas_handle) {

    const float alpha = 1.0f / A.col(); 
    const float beta = 0; 

    int m = C.row();
    int n = C.col();
    int k = at ? A.row() : A.col();
    int repeats = std::max(1e11/(m*k*n), 10.0);

    // warm up
    if (HIPBLAS_STATUS_SUCCESS != hipblasSgemm(cublas_handle,
                at ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                bt ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                m, n, k,
                &alpha,
                A.data(), A.row(),
                B.data(), B.row(),
                &beta,
                C.data(), C.row())) {
        throw std::runtime_error("sgemm failed in warming up!");
    }
    if (hipSuccess != hipDeviceSynchronize()) {
        throw std::runtime_error("device sync error after warming up!");
    }

    // go
    auto t0 = std::chrono::steady_clock::now();
    for (int i=0; i<repeats; i++) {
        hipblasStatus_t stat = hipblasSgemm(cublas_handle,
                    at ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                    bt ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                    m, n, k,
                    &alpha,
                    A.data(), A.row(),
                    B.data(), B.row(),
                    &beta,
                    C.data(), C.row());

        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf("stat: %d\n", stat);
            throw std::runtime_error("sgemm failed!");
        }
    }
    if (hipSuccess != hipDeviceSynchronize()) {
        throw std::runtime_error("device sync error!");
    }
    auto t1 = std::chrono::steady_clock::now();

    return std::chrono::duration<double, std::micro>(t1-t0).count() / repeats;
}




int main()
{
    // initialize
    hipFree(0);
    hipblasHandle_t cublas_handle;
    hipblasStatus_t status = hipblasCreate(&cublas_handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS init failed!\n");
        exit(1);
    }

    hiprandGenerator_t curand_gen;
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);


    FILE *fout = fopen("gemm.csv", "w");
    if (!fout) {
        printf("can't open output file\n");
        exit(1);
    }

    printf("%7s%7s%7s%7s%7s%13s\n", "m", "n", "k", "at", "bt", "time(us)");
    fprintf(fout, "%7s,%7s,%7s,%7s,%7s,%13s\n", "m", "n", "k", "at", "bt", "time(us)");
    int m, n, k;
    while (scanf("%d%d%d", &m, &n, &k) != EOF) {
        bool at, bt;
        char buf[256];
        scanf("%s", buf);
        at = buf[0] == 't';
        scanf("%s", buf);
        bt = buf[0] == 't';


        Matrix<float> a(m, k), 
                      b(at ? m:(bt ? n:k), bt ? k:n), 
                      c(at ? k:m, n); 

        a.fillRand(curand_gen);
        b.fillRand(curand_gen);
        c.fill(0);

        double t = gemm(a, b, c, at, bt, cublas_handle);

        printf("%7d%7d%7d%7c%7c", m, n, k, at ? 't':'n', bt ? 't':'n');
        printf("%13.3lf\n", t);

        fprintf(fout, "%7d,%7d,%7d,%7c,%7c,", m, n, k, at ? 't':'n', bt ? 't':'n');
        fprintf(fout, "%13.3lf\n", t);
    }

    fclose(fout);

    return 0;
}


